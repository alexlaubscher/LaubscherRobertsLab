#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Practice with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024

__global__ void kidKernel(void) {
    for (int i = SIZE - 5; i < SIZE; ++i) {
        printf("c[%d] = %d\n", i, dev_c[i]);
    }
}

__global__ void VectorAdd(float *a, float *b, float *c, int n) {
    printf("Inside the kernel");
    int i = threadIdx.X;

    if (i < n)
        c[i] = a[i] + b[i];

    kidKernel <<<1, 1>>> ();

}


int main() {
    float *dev_a, *dev_b, *dev_c, *dev_d;

    float *a = (float *)calloc(SIZE, sizeof(float));
    float *b = (float *)calloc(SIZE, sizeof(float));
    float *c = (float *)calloc(SIZE, sizeof(float));
    float *d = (float *)calloc(SIZE, sizeof(float));

    hipMalloc((void **) &dev_a, SIZE*sizeof(float));
    hipMalloc((void **) &dev_b, SIZE*sizeof(float));
    hipMalloc((void **) &dev_c, SIZE*sizeof(float));
    hipMalloc((void **) &dev_d, SIZE*sizeof(float));

    hipMemcpy(dev_a, a, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE, hipMemcpyHostToDevice);

    for (int i = 0; i < SIZE; ++i) {
        a[i] = i;
        b[i] = i;
    }

    VectorAdd <<<1, SIZE>>> (&dev_a, &dev_b, &dev_c, SIZE);

    free(a), free(b);
    hipFree(dev_a), hipFree(dev_b), hipFree(dev_c), hipFree(dev_d);
}
