#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Practice with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024

__global__ void kidKernel(void) {
    for (int i = SIZE - 5; i < SIZE; ++i) {
        printf("c[%d] = %d\n", i, dev_c[i]);
    }
}

__global__ void VectorAdd(int *a, int *b, int n) {
    printf("Inside the kernel");
    int i = threadIdx.X;

    if (i < n)
        c[i] = a[i] + b[i];

    kidKernel <<<1, 1>>> ();

    cudaDeviceSychronize();
}


int main() {
    float *dev_a, *dev_b, *dev_c, *dev_d;

    int* a = (float *)calloc(SIZE, sizeof(float));
    int* b = (float *)calloc(SIZE, sizeof(float));
    int* c = (float *)calloc(SIZE, sizeof(float));
    int* d = (float *)calloc(SIZE, sizeof(float));

    hipMalloc((void **) &dev_a, count*sizeof(float));
    hipMalloc((void **) &dev_b, count*sizeof(float));
    hipMalloc((void **) &dev_c, count*sizeof(float));
    hipMalloc((void **) &dev_d, count*sizeof(float));

    hipMemcpy(d_a, a, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE, hipMemcpyHostToDevice);

    for (int i = 0; i < SIZE; ++i) {
        a[i] = i;
        b[i] = i;
    }


    free(a), free(b);
    hipFree(dev_a), hipFree(dev_b), hipFree(dev_c), hipFree(dev_d);
}
