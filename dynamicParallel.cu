#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Practice with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024

__global__ void kidKernel(void) {
    for (int i = SIZE - 5; i < SIZE; ++i) {
        printf("c[%d] = %d\n", i, dev_c[i]);
    }
}

__global__ void VectorAdd(int *a, int *b, int c, int n) {
    printf("Inside the kernel");
    int i = threadIdx.X;

    if (i < n)
        c[i] = a[i] + b[i];

    kidKernel <<<1, 1>>> ();

    cudaDeviceSychronize();
}


int main() {
    float *dev_a, *dev_b, *dev_c, *dev_d;

    int *a = (float *)calloc(SIZE, sizeof(float));
    int *b = (float *)calloc(SIZE, sizeof(float));
    int *c = (float *)calloc(SIZE, sizeof(float));
    int *d = (float *)calloc(SIZE, sizeof(float));

    hipMalloc((void **) &dev_a, SIZE*sizeof(float));
    hipMalloc((void **) &dev_b, SIZE*sizeof(float));
    hipMalloc((void **) &dev_c, SIZE*sizeof(float));
    hipMalloc((void **) &dev_d, SIZE*sizeof(float));

    hipMemcpy(dev_a, a, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE, hipMemcpyHostToDevice);

    for (int i = 0; i < SIZE; ++i) {
        a[i] = i;
        b[i] = i;
    }

    VectorAdd <<<1, SIZE>>> (dev_a, dev_b, dev_c, SIZE);

    free(a), free(b);
    hipFree(dev_a), hipFree(dev_b), hipFree(dev_c), hipFree(dev_d);
}
