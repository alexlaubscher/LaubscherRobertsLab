#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Practice with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int n) {
    printf("Inside the kernel");
    int i = threadIdx.X;

    if (i < n)
        c[i] = a[i] + b[i];
}


int main() {
    float *dev_a, *dev_b, *dev_c, *dev_d;

    a = (float *)calloc(SIZE, sizeof(float));
    b = (float *)calloc(SIZE, sizeof(float));
    c = (float *)calloc(SIZE, sizeof(float));
    d = (float *)calloc(SIZE, sizeof(float));

    hipMalloc((void **) &dev_a, count*sizeof(float));
    hipMalloc((void **) &dev_b, count*sizeof(float));
    hipMalloc((void **) &dev_c, count*sizeof(float));
    hipMalloc((void **) &dev_d, count*sizeof(float));


    for (int i = 0; i < SIZE; ++i) {
        a[i] = i;
        b[i] = i;
    }



    free(a), free(b);
    hipFree(dev_a), hipFree(dev_b), hipFree(dev_c), hipFree(dev_d);
}
