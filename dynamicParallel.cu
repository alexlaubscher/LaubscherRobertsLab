/*
* Alex Laubscher
* Practice with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024

int main() {
    int *a, *b, *c;

    hipMallocManaged(&a, SIZE * sizeof(int));
    hipMallocManaged(&b, SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));
    hipMallocManaged(&d, SIZE * sizeof(int));

    for (int i = 0; i < SIZE; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
        d[i] = 0;
    }

    VectorAdd
}
