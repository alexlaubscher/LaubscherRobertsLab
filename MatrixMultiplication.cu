/*
* Alex Laubscher
* Matrix Multiplication
*/

#include <stdio.h>
#define SIZE 1024

__device__ void multiply(int *a, int *b, int *c, int *a_rows, int *a_cols,
    int *b_rows, int *b_cols) {
    // Will be the index for the product array
    int i = threadIdx.x;
    int block = b_rows * b_cols;
    int multiple = i / block;

    // Will be the index for the left matrix
    int offset = i % block;
    int j = multiple * block + offset;

    // Will be the index for the right matrix
    int rem = i / a_cols;
    int offset = i / (a_rows * a_cols);
    int k = rem * b_cols + offset;

    c[i] = b[j] * c[k];
}

__device__ void sum(int *temp, int *c, int *b_rows) {
    int i = threadIdx.x;
    int index = i / b_rows;
    c[index] += c[i];
}

int main(void) {
    // Initialize the pointers
    int *a, *b, *c;
    int *d_a, *d_b, *d_c, *d_temp;

    // Create the size of the elements
    int a_rows = 3;
    int a_cols = 2;
    int b_rows = 2;
    int b_cols = 4;
    int a_size = a_rows * a_cols * sizeof(int);
    int b_size = b_rows * b_cols * sizeof(int);
    int prod_size = a_rows * b_cols * sizeof(int);
    int temp_size = a_rows * a_cols * b_cols * sizeof(int);

    // Makes sure the matrices can even be multiplied
    if (a_cols != b_rows) {
        printf("Illegal Matrix Sizes\n");
        return 1;
    }

    // Allocate memory on GPU for matrices
    hipMalloc((void **) &d_a, a_size);
    hipMalloc((void **) &d_b, b_size);
    hipMalloc((void **) &d_temp, temp_size);
    hipMalloc((void **) &d_c, prod_size);

    // Allocate memory on the host
    a = (int *)malloc(a_size);
    b = (int *)malloc(b_size);
    c = (int *)malloc(prod_size);

    // Generate the matrices
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Move the matrices data to the GPU
    hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, b_size, hipMemcpyHostToDevice);

    // Initialize the kernel for multiplying the matrices
    multiply<<<1, temp_size>>>(d_a, d_b, d_c, a_rows, a_cols, b_rows, b_cols);

    // Initialize the kernel for summing the resultant matrix
    sum<<<1, temp_size>>>(d_temp, d_c, b_rows);

    // Move the product back to the host
    hipMemcpy(c, d_c, prod_size, hipMemcpyDeviceToHost);

    // Print out the results
    for (int i = 0; i < prod_size; i++) {
        printf("c[%d] = %d\n", i, c[i]);
    }

    // Free the memory
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); hipFree(d_temp);

    return 0;
}
