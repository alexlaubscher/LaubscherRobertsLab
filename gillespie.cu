/*
* Alex Laubscher
* Gillespie Algorithm
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE 1024

int main(void) {
    // Starting the timer
    clock_t time_elapsed = clock();

    // Initializing variables for the while loop
    // double counter;
    int total;
    double tau;
    double sample;

    // Initial population
    int pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 100000;

    // Moved this outside because its going to be constant
    int birth = 1000;
    
    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {
	
        // Sum over the propensities
        total = birth + pop;

        // Calculate time step
        tau = (1.0 / total) * log((double) rand() / (RAND_MAX));

        // Second random choice
        sample = total * ((double) rand() / (RAND_MAX));

        // Update populations based on second urn
        if (sample < birth) {
            pop++;
        } else {
            pop--;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        // counter++;
    }

    // End the time and convert to sec
    time_elapsed = (clock() - time_elapsed);
    double timer = ((double) time_elapsed) / CLOCKS_PER_SEC;

    // Calculate the reactions per sec
    double rate = 200020751 / timer;
    printf("Population: %d\n", pop);
    // printf("Counter: %f\n", counter);
    printf("Timer: %f\n", timer);
    printf("Rate: %f\n", rate);
}
