#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Gillespie Algorithm
*/

#include <stdio.h>
#include <time.h>
#define SIZE 1024

int main(void) {
    // Starting the timer
    clock_t start = clock();

    // Initializing variables for the while loop
    double counter;
    int birth;
    int *death;
    int total;
    double tau;
    double sample;

    // Initial population
    double pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 100000;

    // Moved this outside because its going to be constant
    birth = 1000;
    death = pop;

    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {

        // Sum over the propensities
        total = birth + death;

        // Calculate time step
        tau = (1.0 / total) * log((double) rand() / (RAND_MAX));

        // Second random choice
        sample = total * ((double) rand() / (RAND_MAX));

        // Update populations based on second urn
        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter++;
    }

    // End the time and convert to sec
    clock_t end = clock();
    int timer = (end - start) / CLOCKS_PER_SEC;

    //Calculate the reactions per sec
    double rate = counter / timer;
    printf("Population: %f\n", pop);
    printf("Counter: %f\n", counter);
    printf("Timer: %d\n", timer);
    printf("Rate: %f\n", rate);
}
