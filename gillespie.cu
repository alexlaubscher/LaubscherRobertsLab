/*
* Alex Laubscher
* Gillespie Algorithm
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE 1024

int main(void) {
    // Starting the timer
    clock_t start = clock();

    // Initializing pointers
    int *urn;
    int *d_urn;

    // Initializing variables for the while loop
    double counter;
    int birth;
    int death;
    int total;
    double tau;
    double sample;

    // Initial population
    int pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 1;

    // Allocating memory for the random numbers
    urn = (int *)malloc(SIZE * sizeof(int));
    hipMalloc((void **) &d_urn, SIZE * sizeof(int));

    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {
        // Setting the propensity of the rxn
        birth = 1000;
        death = pop;

        // Sum over the propensities
        total = birth + death;

        // Calculate time step
        tau = (1.0 / total) * log((rand() % 10000) / 10000.0);

        // Second random choice
        sample = total * (rand() % 10000) / 10000.0;

        // Update populations based on second urn
        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter = counter + 1;
	printf("%f -- %f\n", tau, sample);
    }

    // End the time and convert to sec
    // clock_t end = clock();
    // int timer = (end - start) / CLOCKS_PER_SEC * 1000.0;

    // Calculate the reactions per sec
    // double rate = counter / timer;
    // printf("%d", timer);
}
