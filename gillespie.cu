#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Gillespie Algorithm
*/

#include <stdio.h>
#define SIZE 1024

int main(void) {
    // Starting the timer
    clock_t start = clock();

    // Initializing pointers
    int *urn;
    int *d_urn;

    // Initializing variables for the while loop
    int counter;
    int birth;
    int death;
    int total;
    int tau;
    int sample;

    // Initial population
    int pop = 0;

    // Initializing time
    int time = 0;
    int maxTime = 1000;

    // Allocating memory for the random numbers
    urn = (int *)malloc(SIZE * sizeof(int));
    d_urn = hipMalloc((void **) &d_urn, SIZE * sizeof(int));

    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {
        // Setting the propensity of the rxn
        birth = 1000;
        death = pop;

        // Sum over the propensities
        total = birth + death;

        // Calculate time step
        tau = 1 / total * log(rand());

        // Second random choice
        sample = total * rand();

        // Update populations based on second urn
        if (sample < birth) {
            y1 += 1;
        } else {
            y1 -= 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter++;
    }

    // End the time and convert to sec
    clock_t end = clock();
    double time = (double) (end - start) / CLOCKS_PER_SEC * 1000.0;

    // Calculate the reactions per sec
     double rate = counter / time;
    printf("%d", rate);
}
