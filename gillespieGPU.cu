/*
* Alex Laubscher
* Gillespie Algorithm
*/


#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>
#define SIZE 1024

int main(void) {

    // Initializing variables for gillespie algorithm
    double counter;
    int death;
    int total;
    double tau;
    double sample;

    // Initialize variables for the GPU generator
    int count = 50000;
    hiprandGenerator_t gen;
    float *devURN;
    float *hostURN;

    // Allocate n floats on host
    hostURN = (float *)calloc(count, sizeof(float));

    // Allocate n floats on device
    hipMalloc((void **) &devURN, count*sizeof(float));

    // Create the generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Initial population
    double pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 100000;

    // Can be outside cuz it never changes
    int birth = 1000;

    // Starting the timer
    clock_t time_elapsed = clock();

    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {
        // Setting the propensity of the rxn
        death = pop;

        // Sum over the propensities
        total = birth + death;

        // Need to cast the double
        int check = (int) counter % count;

        if (check == 0) {
            // Generate the floats
            hiprandGenerateUniform(gen, devURN, count);

            // Copy the numbers back to the device
            hipMemcpy(hostURN, devURN, count*sizeof(float),
                hipMemcpyDeviceToHost);
        }

        // Calculate time step
        tau = (1.0 / total) * log(hostURN[check * 2]);

        // Second random choice
        sample = total * (hostURN[check * 2 + 1]);

        // Update populations based on second urn
        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter++;
    }

    // End the time and convert to sec
    time_elapsed = clock() - time_elapsed;
    double timer = ((double) time_elapsed) / CLOCKS_PER_SEC;

    //Calculate the reactions per sec
    double rate = counter / timer;
    printf("Population: %f\n", pop);
    printf("Counter: %f\n", counter);
    printf("Timer: %f\n", timer);
    printf("Rate: %f\n", rate);

    hiprandDestroyGenerator(gen);
    hipFree(devURN);
    free(hostURN);

    return 0;
}
