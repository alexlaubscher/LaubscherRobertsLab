/*
* Alex Laubscher
* Gillespie Algorithm
* Uses a GPU generator for the numbers
*/


#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdio.h>
#include <time.h>

int main() {
    // Starting the timer
    clock_t time_elapsed = clock();

    // Initializing variables for gillespie algorithm
    int counter;
    int death;
    int total;
    double tau;
    double sample;
    int check;

    // Initialize variables for the GPU generator
    int count = 2500000;
    hiprandGenerator_t gen;
    float *devURN;
    float *hostURN;

    // Allocate n floats on host
    hostURN = (float *)calloc(count, sizeof(float));

    // Allocate n floats on device
    hipMalloc((void **) &devURN, count*sizeof(float));

    // Create the generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Initial population
    int pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 100000;

    // Can be outside cuz it never changes
    int birth = 1000;

    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {
        // Setting the propensity of the rxn
        death = pop;

        // Sum over the propensities
        total = birth + death;

        // Need to cast the double
        check = counter % (count / 2);

        if (check == 0) {
            // Generate the floats
            hiprandGenerateUniform(gen, devURN, count);

            // Copy the numbers back to the device
            hipMemcpy(hostURN, devURN, count*sizeof(float),
                hipMemcpyDeviceToHost);
        }

        // Calculate time step
        tau = (1.0 / total) * log(hostURN[check * 2]);

        // Second random choice
        sample = total * (hostURN[check * 2 + 1]);

        // Update populations based on second urn
        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter++;
    }

    // End the time and convert to sec
    time_elapsed = clock() - time_elapsed;
    double timer = ((double) time_elapsed) / CLOCKS_PER_SEC;

    //Calculate the reactions per sec
    double rate = counter / timer;
    printf("Population: %f\n", pop);
    printf("Counter: %d\n", counter);
    printf("Timer: %f\n", timer);
    printf("Rate: %f\n", rate);

    hiprandDestroyGenerator(gen);
    hipFree(devURN);
    free(hostURN);

    return 0;
}
