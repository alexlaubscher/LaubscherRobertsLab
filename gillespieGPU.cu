#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Gillespie Algorithm
*/

#include <stdio.h>
#include <time.h>
#define SIZE 1024

__device__ void genUrn(double *urn) {
    urn[threadIdx.x] = (double) rand() / (RAND_MAX);
}

int main(void) {
    // Starting the timer
    clock_t start = clock();

    // Initializing pointers
    double *urn;
    double *d_urn;

    // Initializing variables for the while loop
    double counter;
    int birth;
    int death;
    int total;
    double tau;
    double sample;
    int allocSize = SIZE * sizeof(double);

    // Initial population
    double pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 100000;

    // Allocating memory for the random numbers
    urn = (double *)malloc(allocSize);
    hipMalloc((void **) &d_urn, allocSize);

    // Run the while loop over 100,000 simulation seconds
    while (time < maxTime) {
        // Setting the propensity of the rxn
        birth = 1000;
        death = pop;

        // Sum over the propensities
        total = birth + death;

        // Need to cast the double
        int check = counter % 512

        if (check == 0) {
            genUrn<<<1, SIZE>>>(d_urn);
            hipMemcpy(urn, d_urn, allocSize, hipMemcpyDeviceToHost);
        }

        // Calculate time step
        tau = (1.0 / total) * log(urn[check * 2]);

        // Second random choice
        sample = total * (urn[check * 2 + 1]);

        // Update populations based on second urn
        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter++;
    }

    // End the time and convert to sec
    clock_t end = clock();
    int timer = (end - start) / CLOCKS_PER_SEC;

    //Calculate the reactions per sec
    double rate = counter / timer;
    printf("Population: %f\n", pop);
    printf("Counter: %f\n", counter);
    printf("Timer: %d\n", timer);
    printf("Rate: %f\n", rate);
}
