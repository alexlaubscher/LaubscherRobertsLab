#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Gillespie Algorithm
* Runs a singular simulation on a GPU
*/

#include <hiprand.h>
#include <stdio.h>
#include <time.h>

__global__ void simulation(int count, float *tauURN, float *distURN, hiprandGenerator_t gen) {

    // Same initialization of variables
    int counter;
    int death;
    int total;
    double tau;
    double sample;
    int check;

    // Initial population
    int pop = 0;

    // Initializing time
    double time = 0;
    double maxTime = 100000;

    // Birth rate
    int birth = 1000;

    // Start the timer
    clock_t time_elapsed = clock();

    // Body of the gillespie
    while (time < maxTime) {
        // Setting the propensity
        death = pop;

        // Sum over the propensities
        total = birth + death;

        // Check if array is empty
        check = counter % count;

        if (check == 0) {
            // Generate the new arrays
            hiprandGenerateUniform(gen, tauURN, count);
            hiprandGenerateUniform(gen, distURN, count);
        }

        // Gives us the time step
        tau = (1.0 / total) * tauURN[check];

        // Second random choice
        sample = total * distURN[check];

        // Update populations
        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        // Update the time step
        time = time - tau;

        // Increment the counter
        counter++;
    }

    // Calculate the time elapsed
    time_elapsed = clock() - time_elapsed;
    double timer = ((double) time_elapsed) / CLOCKS_PER_SEC;

    //Calculate the reactions per sec
    double rate = counter / timer;
    printf("Population: %f\n", pop);
    printf("Counter: %d\n", counter);
    printf("Timer: %f\n", timer);
    printf("Rate: %f\n", rate);

}

int main() {
    // Initialize streams
    hipStream_t stream1, stream2, stream3, stream4, stream5;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    hipStreamCreate(&stream5);

    // Create the generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    int count = 2500000;
    float *tauURN;
    float *distURN;
    hipMalloc((void **) &tauURN, count*sizeof(float));
    hipMalloc((void **) &distURN, count*sizeof(float));

    // Run a single simulation on the device
    simulation<<<1, 1024, 0, stream1>>>(count, tauURN, distURN, gen);


    // FREE HOSTS
    hipHostFree(;alkdf;alkj);

    hiprandDestroyGenerator(gen);
    hipFree(tauURN);
    hipFree(distURN);
    return 0;
}
