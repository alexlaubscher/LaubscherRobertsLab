#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Gillespie Algorithm
* Runs a singular simulation on a GPU
*/

#include <hiprand.h>
#include <stdio.h>
#include <time.h>

__device__ void simulation() {

}

int main() {
    // Start the timer 
    clock_t time_elapsed = clock();

    // Run a single simulation on the device
    simulation<<<1, 1>>>();

    // Calculate the time elapsed
    time_elapsed = clock() - time_elapsed;
    double timer = ((double) time_elapsed) / CLOCKS_PER_SEC;

    return 0;
}
