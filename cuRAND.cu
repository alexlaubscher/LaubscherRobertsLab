#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Random Number Generation
*/

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
    // Initialize variables
    int count = 500000;
    int i;
    hiprandGenerator_t gen;
    float *devURN;
    float *hostURN;
    float *cpuURN;

    // Allocate n floats on host
    hostURN = (float *)calloc(count, sizeof(float));
    cpuURN = (float*)calloc(count, sizeof(float));

    // Allocate n floats on device
    hipMalloc((void **) &devURN, count*sizeof(float));

    // Create the generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    clock_t time_elapsed = clock();
    
    // Generate the floats
    hiprandGenerateUniform(gen, devURN, count);

    // Copy the numbers back to the device
    hipMemcpy(hostURN, devURN, count*sizeof(float), hipMemcpyDeviceToHost);

    time_elapsed = (clock() - time_elapsed);
    double GPU_time = ((double) time_elapsed) / CLOCKS_PER_SEC;

    time_elapsed = clock();

    for (i = 0; i < count; i++) {
        cpuURN[i] = rand();
    }

    time_elapsed = (clock() - time_elapsed);
    double CPU_time = ((double) time_elapsed) / CLOCKS_PER_SEC;

    for (i = 0; i < 10; i++) {
        printf("GPU: %f CPU: %f\n", hostURN[i], cpuURN[i]);
    }

    printf("GPU time: %f\n", GPU_time);
    printf("CPU time: %f\n", CPU_time);

    hiprandDestroyGenerator(gen);
    hipFree(devURN);
    free(cpuURN);
    free(hostURN);
}
