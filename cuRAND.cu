#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Random Number Generation
*/

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

int main() {
    // Initialize variables
    int count = 1000;
    int i;
    hiprandGenerator_t gen;
    float *devURN;
    float *hostURN;
    float *cpuURN;

    // Allocate n floats on host
    hostURN = (float *)calloc(count, sizeof(float));
    cpuURN = (float*)calloc(count, sizeof(float));

    // Allocate n floats on device
    hipMalloc((void **) &devURN, count*sizeof(float));

    // Create the generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    clock_t startGPU = clock();
        // Generate the floats
        hiprandGenerateUniform(gen, devURN, count);

        // Copy the numbers back to the device
        hipMemcpy(devURN, hostURN, count*sizeof(float), hipMemcpyDeviceToHost);
    double timeGPU = (clock() - startGPU) / CLOCKS_PER_SEC;

    clock_t startCPU = clock();
        for (i = 0; i < count; i++) {
            cpuURN[i] = rand();
        }
    double timeCPU = (clock() - startCPU) / CLOCKS_PER_SEC;

    printf("GPU time: %f\n", timeGPU);
    printf("CPU time: %f\n", timeCPU);

    hiprandDestroyGenerator(gen);
    hipFree(devURN);
    free(cpuURN);
    free(hostURN);
}
