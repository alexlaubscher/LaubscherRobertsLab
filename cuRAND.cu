#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Random Number Generation
*/

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>

int main() {
    // Initialize variables
    int count = 1000;
    int i;
    hiprandGenerator_t gen;
    float *devURN;
    float *hostURN;
    float *cpuURN;

    // Allocate n floats on host
    hostURN = (float *)calloc(count, sizeof(float));
    cpuURN = (float*)calloc(count, sizeof(float));

    // Allocate n floats on device
    hipMalloc((void **) &devURN, count*sizeof(float));

    // Create the generator
    hiprandCreateGenerator(&gen, CURAND_RNG_PSEUDO_DEAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    clock_t startGPU = clock();
        // Generate the floats
        hiprandGenerateUniform(gen, devURN, count);

        // Copy the numbers back to the device
        hipMemcpy(devURN, hostURN, count*sizeof(float), hipMemcpyDeviceToHost);
    double timeGPU = (clock() - startGPU) / CLOCKS_PER_SEC;

    clock_t startCPU = clock();
        for (i = 0; i < count; i++) {
            cpuURN[i] = rand();
        }
    double timeCPU = (clock() - startCPU) / CLOCKS_PER_SEC;

    printf("GPU time: %f\n", timeGPU);
    printf("CPU time: %f\n", timeCPU);

    cudaDestroyGenerator(gen);
    hipFree(devURN);
    free(cpuURN);
    free(hostURN);
}
