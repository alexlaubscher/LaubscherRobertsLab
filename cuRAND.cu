#include "hip/hip_runtime.h"
/*
* Alex Laubscher
* Random Number Generation
*/

#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
    // Initialize variables
    int count = 10000;
    int i;
    hiprandGenerator_t gen;
    float *devURN;
    float *hostURN;
    float *cpuURN;

    // Allocate n floats on host
    hostURN = (float *)calloc(count, sizeof(float));
    cpuURN = (float*)calloc(count, sizeof(float));

    // Allocate n floats on device
    hipMalloc((void **) &devURN, count*sizeof(float));

    // Create the generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    clock_t startGPU = clock();
        // Generate the floats
        hiprandGenerateUniform(gen, devURN, count);

        // Copy the numbers back to the device
        hipMemcpy(devURN, hostURN, count*sizeof(float), hipMemcpyDeviceToHost);
    double timeGPU = (clock() - startGPU) / CLOCKS_PER_SEC;

    clock_t startCPU = clock();
        for (i = 0; i < count; i++) {
            cpuURN[i] = rand();
        }
    double timeCPU = (clock() - startCPU) / CLOCKS_PER_SEC;

    printf("GPU time: %f\n", timeGPU);
    printf("CPU time: %f\n", timeCPU);

    hiprandDestroyGenerator(gen);
    hipFree(devURN);
    free(cpuURN);
    free(hostURN);
}
