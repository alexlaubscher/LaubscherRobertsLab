/*
* Alex Laubscher
* Gillespie with Dynamic Parallelism
*/


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

__device__ float *normURN;
__device__ float *logURN;
__device__ float *normURN2;
__device__ float *logURN2;

__global__ void genURN(float *normURN, int *count) {
    int i = threadIdx.x;

    if (i < *count) {
        hiprandState state;
        hiprand_init(clock64(), i, 0, &state);
        normURN[i] = hiprand_uniform(&state);
    }
}

__global__ void genLogURN(float *logURN, int *count) {
    int i = threadIdx.x;

    if (i < *count) {
        hiprandState state;
        hiprand_init(clock64(), i, 0, &state);
        normURN[i] = log(hiprand_uniform(&state));
    }
}

__global__ void devMain(int *counter, int *death, int *total, double *tau,
    double *sample, int *check, int *count, int *pop, double *time,
    double *maxTime, int *birth, int *swap) {

    hipMalloc((void **) &normURN, 250000*sizeof(float));
    hipMalloc((void **) &logURN, 250000*sizeof(float));
    hipMalloc((void **) &normURN2, 250000*sizeof(float));
    hipMalloc((void **) &logURN2, 250000*sizeof(float));

    *count = 250000;
    *pop = 0;
    *time = 0;
    *maxTime = 10;
    *birth = 1000;

    while(time < maxTime) {
        printf("%f < %f\n", *time, *maxTime);
        *death = *pop;

        *total = *birth + *death;

        *check = *counter % (*count);
        genURN<<<1, 512>>>(logURN, count);
        genLogURN<<<1, 512>>>(normURN, count);

        if (*check == 0) {
            if (*swap == 1) {
                genURN<<<1, 512>>>(logURN2, count);
                genLogURN<<<1, 512>>>(normURN2, count);
                *swap = 2;
            } else {
                genURN<<<1, 512>>>(logURN, count);
                genLogURN<<<1, 512>>>(normURN, count);
                *swap = 1;
            }
        }

        if (*swap == 1) {
            *tau = (1.0 / *total) * logURN[*check];
            *sample = *total * normURN[*check];
        } else {
            *tau = (1.0 / *total) * logURN2[*check];
            *sample = *total * normURN2[*check];
        }


        if (*sample < *birth) {
            *pop = *pop + 1;
        } else {
            *pop = *pop - 1;
        }

        *time = *time - *tau;

        *counter++;
    }

    printf("something is wrong here\n");

    hipFree(normURN);
    hipFree(logURN);
    hipFree(normURN2);
    hipFree(logURN2);

    printf("Population: %f\n", *pop);
    printf("Counter: %d\n", *counter);
}

int main(void) {
    int *counter;
    int *death;
    int *total;
    double *tau;
    double *sample;
    int *check;
    int *count;
    int *pop;
    double *time;
    double *maxTime;
    int *birth;
    int *swap;

    hipMalloc(&counter, sizeof(int));
    hipMalloc(&death, sizeof(int));
    hipMalloc(&total, sizeof(int));
    hipMalloc(&tau, sizeof(double));
    hipMalloc(&sample, sizeof(double));
    hipMalloc(&check, sizeof(int));
    hipMalloc(&count, sizeof(int));
    hipMalloc(&pop, sizeof(int));
    hipMalloc(&time, sizeof(double));
    hipMalloc(&maxTime, sizeof(double));
    hipMalloc(&birth, sizeof(int));
    hipMalloc(&swap, sizeof(int));

    devMain<<<1, 128>>>(counter, death, total, tau, sample, check, count,
        pop, time, maxTime, birth, swap);

    hipFree(counter);
    hipFree(death);
    hipFree(total);
    hipFree(tau);
    hipFree(sample);
    hipFree(check);
    hipFree(pop);
    hipFree(time);
    hipFree(maxTime);
    hipFree(birth);
    hipFree(swap);
}
