/*
* Alex Laubscher
* Gillespie with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
    int *counter;
    int *death;
    int *total;
    double *tau;
    double *sample;
    int *check;
    int *pop;
    double *time;
    double *maxTime;
    int *birth;
    double *normURN;
    double *logURN;

    hipMalloc(&counter, size*sizeof(int));
    hipMalloc(&death, size*sizeof(int));
    hipMalloc(&total, size*sizeof(int));
    hipMalloc(&tau, size*sizeof(double));
    hipMalloc(&sample, size*sizeof(double));
    hipMalloc(&check, size*sizeof(int));
    hipMalloc(&pop, size*sizeof(int));
    hipMalloc(&time, size*sizeof(double));
    hipMalloc(&maxTime, size*sizeof(double));
    hipMalloc(&birth, size*sizeof(int));
    hipMalloc((void **) &normURN, 250000*sizeof(double));
    hipMalloc((void **) &logURN, 250000*sizeof(double));

    devMain<<<1, 128>>>(counter, death, total, tau, sample, check
        pop, time, maxTime, birth, normURN, logURN);

    hipFree(counter);
    hipFree(death);
    hipFree(total);
    hipFree(tau);
    hipFree(sample);
    hipFree(check);
    hipFree(pop);
    hipFree(time);
    hipFree(maxTime);
    hipFree(birth);
    hipFree(normURN);
    hipFree(logURN);
}
