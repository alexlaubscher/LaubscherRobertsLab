/*
* Alex Laubscher
* Gillespie with Dynamic Parallelism
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>

__device__ void genURN() {

}

__device__ void genLogURN() {

}

__device__ void devMain(int *counter, int *death, int *total, double *tau,
    double *sample, int *check, int *pop, double *time, double *maxTime,
    int *birth, double *normURN, double *logURN) {
    count = 250000;
    pop = 0;
    time = 0;
    maxTime = 100000;
    birth = 1000;

    while(time < maxTime) {
        death = pop;

        total = birth + death;

        check = counter % (count/2);

        // if (check == 0) {
        //     genURN<<<1, 512>>>();
        //     genLogURN<<<1, 512>>>();
        // }

        tau = (1.0 / total) * logURN[check];

        sample = total * normURN[check];

        if (sample < birth) {
            pop = pop + 1;
        } else {
            pop = pop - 1;
        }

        time = time - tau;

        counter++;
    }
}

int main() {
    int *counter;
    int *death;
    int *total;
    double *tau;
    double *sample;
    int *check;
    int *pop;
    double *time;
    double *maxTime;
    int *birth;
    double *normURN;
    double *logURN;

    hipMalloc(&counter, size*sizeof(int));
    hipMalloc(&death, size*sizeof(int));
    hipMalloc(&total, size*sizeof(int));
    hipMalloc(&tau, size*sizeof(double));
    hipMalloc(&sample, size*sizeof(double));
    hipMalloc(&check, size*sizeof(int));
    hipMalloc(&pop, size*sizeof(int));
    hipMalloc(&time, size*sizeof(double));
    hipMalloc(&maxTime, size*sizeof(double));
    hipMalloc(&birth, size*sizeof(int));
    hipMalloc((void **) &normURN, 250000*sizeof(double));
    hipMalloc((void **) &logURN, 250000*sizeof(double));

    devMain<<<1, 128>>>(counter, death, total, tau, sample, check
        pop, time, maxTime, birth, normURN, logURN);

    hipFree(counter);
    hipFree(death);
    hipFree(total);
    hipFree(tau);
    hipFree(sample);
    hipFree(check);
    hipFree(pop);
    hipFree(time);
    hipFree(maxTime);
    hipFree(birth);
    hipFree(normURN);
    hipFree(logURN);
}
